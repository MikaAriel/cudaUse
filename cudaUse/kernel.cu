#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void kernel(int* data, int size)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	data[id] = id;
}

extern "C"
bool runKernel(int* data, int size)
{
	int* d_data;
	hipMalloc((void**)&d_data, size * sizeof(int));

	hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyHostToDevice);
	kernel << < 1, size >> >(d_data, size);

	hipMemcpy(data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_data);
	return true;
}